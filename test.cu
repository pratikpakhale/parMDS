#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    if (err != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(err));
        return 1;
    }
    printf("Found %d CUDA devices\n", deviceCount);
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s\n", i, prop.name);
    }
    return 0;
}